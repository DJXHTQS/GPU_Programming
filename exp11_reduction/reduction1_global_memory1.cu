#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include"error_check.h"
#include"gpu_timer.h"

#define DTYPE double
#define DTYPE_FORMAT "%lf"
#define BLOCK_SIZE 32

float time_cost_gpu = -1, time_cost_cpu = -1;
hipEvent_t gpu_start, gpu_stop, cpu_start, cpu_stop;

/* CPU implementation */
DTYPE partialSum(DTYPE *vector, int n) {
	DTYPE temp = 0;
	for (int i = 0; i < n; i++) {
		temp += vector[i];
	}
	return temp;
}

/*
 * Todo:
 * reduction kernel in which the threads are mapped to data with stride 2
*/
__global__ void kernel_reduction_non_consecutive(DTYPE *input, DTYPE *output, int n) {
	int tid = threadIdx.x, offset = blockIdx.x*blockDim.x;
	for(int s = 1; s < blockDim.x && tid*2 + s < BLOCK_SIZE; s<<=1){ //主要防无关thread多加
		input[offset+tid*2] += input[offset+tid*2+s];
		__syncthreads();
	}
	if(tid == 0)
		output[blockIdx.x] = input[offset];	
}

/*
 * Todo:
 * reduction kernel in which the threads are consecutively mapped to data
*/
__global__ void kernel_reduction_consecutive(DTYPE *input, DTYPE *output, int n) {
	int tid = threadIdx.x, offset = blockIdx.x*blockDim.x;
	for(int s = BLOCK_SIZE/2; s >= 1 && tid+s < BLOCK_SIZE; s>>=1){
		input[offset+tid] += input[offset+tid+s];
		__syncthreads();
	}
	if(tid == 0)
		output[blockIdx.x] = input[offset];
}

/*
 * Todo:
 * Wrapper function that utilizes cpu computation to sum the reduced results from blocks
*/
DTYPE gpu_reduction_cpu(DTYPE *input, int n,
		void (*kernel)(DTYPE *input, DTYPE *output, int n)) {
	int MEM_SIZE = sizeof(DTYPE) * n;
	DTYPE *in = nullptr, *out = nullptr, *output = nullptr;

	CHECK(hipMalloc((void**)&in, MEM_SIZE));
	CHECK(hipMalloc((void**)&out, MEM_SIZE));
	output = (DTYPE*)malloc(MEM_SIZE);
	CHECK(hipMemcpy(in, input, MEM_SIZE, hipMemcpyHostToDevice));
	int grid = ceil((double)n/BLOCK_SIZE);

	hipEventRecord(gpu_start);
	kernel<<<grid, BLOCK_SIZE>>>(in, out, n);
	hipEventRecord(gpu_stop);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&time_cost_gpu, gpu_start, gpu_stop);
	
	CHECK(hipMemcpy(output, out, MEM_SIZE, hipMemcpyDeviceToHost));
	CHECK(hipFree(in));
	CHECK(hipFree(out));
	
	DTYPE sum = 0;
	for(int i = 0; i < grid; i += 1){
		sum += output[i];
	} 
	free(output);
	return sum;
}


DTYPE* test_data_gen(int n) {
	srand(time(0));
	DTYPE *data = (DTYPE *) malloc(n * sizeof(DTYPE));
	for (int i = 0; i < n; i++) {
		data[i] = 1.0 * (rand() % RAND_MAX) / RAND_MAX;
	}
	return data;
}

void test(int n,
		DTYPE (*reduction)(DTYPE *input, int n,
		                        void (*kernel)(DTYPE *input, DTYPE *output, int n)),
		                        void (*kernel)(DTYPE *input, DTYPE *output, int n))
{
	DTYPE computed_result, computed_result_gpu;
	DTYPE *vector_input;
	vector_input = test_data_gen(n);

	printf("---------------------------\n");


	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventCreate(&cpu_start);
	hipEventCreate(&cpu_stop);

	///cpu
	hipEventRecord(cpu_start);
	computed_result = partialSum(vector_input, n);
	hipEventRecord(cpu_stop);
	hipEventSynchronize(cpu_stop);
	hipEventElapsedTime(&time_cost_cpu, cpu_start, cpu_stop);
	printf("Time cost (CPU):%f ms \n", time_cost_cpu);
	///
	
	///gpu
	computed_result_gpu = reduction(vector_input, n, kernel);
	printf("Time cost (GPU):%f ms \n", time_cost_gpu);
	///
	printf("[%d] Computed sum (CPU): ", n);
	printf(DTYPE_FORMAT, computed_result);
	printf("  GPU result:");
	printf(DTYPE_FORMAT, computed_result_gpu);

	if (abs(computed_result_gpu - computed_result) < 1e-3) {
		printf("  PASSED! \n");
	} else {
		printf("  FAILED! \n");
	}
	printf("\n");

	free(vector_input);

}

int main(int argc, char **argv) {

	int n_arr[] = {1, 7, 585, 5000, 300001, 1<<20};
	for(int i=0; i<sizeof(n_arr)/sizeof(int); i++)
	{
		test(n_arr[i], gpu_reduction_cpu, kernel_reduction_non_consecutive);
		test(n_arr[i], gpu_reduction_cpu, kernel_reduction_consecutive);
	}

	return 0;
}