
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
using namespace std;
int main() {
    int dCount;
    hipGetDeviceCount(&dCount);
    for(int i=0; i<dCount+3; i++)
    {
        hipDeviceProp_t prop;
        hipError_t err = hipGetDeviceProperties(&prop, i);
        if(err != hipSuccess)
          cout<<"yes"<<endl;
        printf("CUDA Device#%d\n", i);
        printf("Device name:%s\n", prop.name);
        printf("multiProcessorCount:%d\n", prop.multiProcessorCount);
        printf("maxThreadsPerBlock:%d\n", prop.maxThreadsPerBlock);
        printf("warpSize:%d\n", prop.warpSize);
        printf("maxThreadsDim[3]:%d, %d, %d\n", 
        prop.maxThreadsDim[0], 
        prop.maxThreadsDim[1], 
        prop.maxThreadsDim[2]);
        printf("maxGridSize[3]:%d, %d, %d\n", 
        prop.maxGridSize[0], 
        prop.maxGridSize[1], 
        prop.maxGridSize[2]);
    }
    return 0;
}
