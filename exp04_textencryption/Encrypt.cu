#include "hip/hip_runtime.h"
#include"error_check.h"
#include"text_helper.h"
#include<stdio.h>

__global__ void encrypt_gpu(char *d_encryptedStr, char *d_decryptedStr, int lenStr, int pwd)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<lenStr)
    {
        d_decryptedStr[idx] = d_encryptedStr[idx] + (idx%pwd+1);
    }
}

void print_msg(char *input, int n)
{
    for(int i=0; i<n; i++)
    {
        printf("%c", input[i]);
    }
}

int main(int argc, char *argv[])
{
    if(argc!=4)
    {
        printf("Usage: command  input-text-file-path  output-text-file-path password");
        return -1;
    }  
    const char *input_file = argv[1];   // "input.txt";
    const char *output_file = argv[2];   // "output.txt";
    const int pwd = atoi(argv[3]);

    printf("\nReading content from %s ...\n", input_file);
    int string_size, read_size;
    char *inputStr = ReadFile(input_file, &read_size, &string_size);
    int lenStr = read_size+1;
    
    //ToDo
    char *outputStr = (char*)malloc(sizeof(char)*lenStr);
    
    char *d_enStr, *d_deStr;
    hipMalloc((void**)&d_enStr, lenStr);
    hipMalloc((void**)&d_deStr, lenStr);

    hipMemcpy(d_enStr, inputStr, lenStr, hipMemcpyHostToDevice);
    encrypt_gpu<<<ceil(lenStr/1024.0), 1024>>>(d_enStr, d_deStr, lenStr, pwd);
    hipMemcpy(outputStr, d_deStr, lenStr, hipMemcpyDeviceToHost);
    hipFree(d_enStr);
    hipFree(d_deStr);

    // Write to output file
    WriteFile(output_file, outputStr, read_size);
    return 0;
}
