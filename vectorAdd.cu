
#include <hip/hip_runtime.h>
#include<iostream>
#include<cmath>
using namespace std;
const int MAX = 100;

template<class T>
__global__
void vecAddKernel(T *A, T *B, T *C, int n){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n) C[i] = A[i]+B[i];
}
template<class T>
void vecAdd(T *A, T *B, T *C, int n){
    int size = n * sizeof(T);
    T *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&d_C, size);

    vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipFree(d_A);hipFree(d_B);hipFree(d_C);
}
int main(){
    int n; cin>>n;
    int A[MAX], B[MAX], C[MAX];
    for(int i = 0; i < n; i++)
        cin >> A[i];
    for(int i = 0; i < n; i++)
        cin >> B[i];
    vecAdd<int>(A, B, C, n);
    for(int i = 0; i < n; i++)
        cout<<C[i]<<' ';
    cout<<endl;
    return 0;
}
